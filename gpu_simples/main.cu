
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>



typedef struct matriz{
    int x;
    int y;
    float *vetor;
    float *cuda_vetor;
    int num_de_pesos;


} TMATRIZ;



TMATRIZ  ler_matriz(char* nome_arq, int eh_entrada){
    FILE *f = fopen(nome_arq,"r");

    TMATRIZ my_matriz;
    if (eh_entrada){
        int x,y,num_de_pesos;
        fscanf(f,"%d %d %d\n",&x,&y,&num_de_pesos );
        my_matriz.x = x;
        my_matriz.y = y;
        my_matriz.num_de_pesos = num_de_pesos;
    }
    else{

         int x,y;
        fscanf(f,"%d %d\n",&x,&y );
        
        my_matriz.x = x;
        my_matriz.y = y;
        
    }

    my_matriz.vetor = (float*) malloc(sizeof(float)* my_matriz.x * my_matriz.y);
    
    for (int i = 0 ; i <  my_matriz.x * my_matriz.y; i++ )
        fscanf(f, "%f",  &my_matriz.vetor[i]);
    fclose(f);
    hipMalloc(&(my_matriz.cuda_vetor), sizeof(float)* my_matriz.x * my_matriz.y );
    hipMemcpy(my_matriz.cuda_vetor,my_matriz.vetor,sizeof(float) * my_matriz.x * my_matriz.y, hipMemcpyHostToDevice);
    

    return my_matriz;
}

#include <math.h>

__device__
float sigmoid(float x) {
     return 1 / (1 + exp(-x));
}





__global__ void mm_kernel(float *mat1, float *mat2, float *mat3, int row1,int col1,int row2,int col2){  
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    float sum = 0;
     if ((i >= row1) || (j >= col2)){
        return;
    }

    for(int k=0 ;k < col1;k++){
        
        sum+=mat1[i*col1 + k ] * mat2[k*col2+j];
    }
    mat3[i * col1+j] = sigmoid(sum);
    

}





int main(){
    TMATRIZ entrada = ler_matriz("input/entrada", 1);
    char file_name[] = "input/peso-0";
    
    
    TMATRIZ *pesos = (TMATRIZ*) malloc(sizeof(TMATRIZ) * entrada.num_de_pesos);
    for(int i = 0; i < entrada.num_de_pesos; i++){
        pesos[i] = ler_matriz(file_name, 0);
        file_name[11]+=1;

    }
    float * matriz_atual = entrada.cuda_vetor;
    int x_atual = entrada.x;
    int y_atual = entrada.y;

    dim3 threads_per_block(32, 32);
    dim3 blocks_per_grid(  1, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    for (int i = 0 ; i < entrada.num_de_pesos;i++){

        float *temp;
        hipMalloc(&temp, sizeof(float) * x_atual * pesos[i].y);
        blocks_per_grid.x =  std::ceil(  (float)x_atual / 32);
        // printf("%d\n", blocks_per_grid.y);
        blocks_per_grid.y = std::ceil( (float)pesos[i].y / 32);
        // teste_kernel<<<blocks_per_grid,threads_per_block>>>();
        mm_kernel<<<blocks_per_grid,threads_per_block>>>(matriz_atual, pesos[i].cuda_vetor, temp,x_atual, y_atual,y_atual, pesos[i].y);
        // float *temp_matriz = multiplica_matriz(matriz_atual, pesos[i].vetor, x_atual, y_atual, pesos[i].y);
        hipDeviceSynchronize();
        hipFree(matriz_atual);
        matriz_atual = temp;
        hipFree(pesos[i].cuda_vetor);
        
        
        y_atual = pesos[i].y;
    }

    hipFree(matriz_atual);
    
    hipEventRecord(stop);
    float milisegundos;
    hipEventElapsedTime(&milisegundos, start,stop);
    FILE *f = fopen( "gpu_otimizado.output" ,"a");
    fprintf(f, "%f\n", milisegundos);
    fclose(f);





    
    
}