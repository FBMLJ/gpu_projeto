
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define BLOCK_DIM 32


typedef struct matriz{
    int x;
    int y;
    float *vetor;
    float *cuda_vetor;
    int num_de_pesos;


} TMATRIZ;



TMATRIZ  ler_matriz(char* nome_arq, int eh_entrada){
    FILE *f = fopen(nome_arq,"r");

    TMATRIZ my_matriz;
    if (eh_entrada){
        int x,y,num_de_pesos;
        fscanf(f,"%d %d %d\n",&x,&y,&num_de_pesos );
        my_matriz.x = x;
        my_matriz.y = y;
        my_matriz.num_de_pesos = num_de_pesos;
    }
    else{

         int x,y;
        fscanf(f,"%d %d\n",&x,&y );
        
        my_matriz.x = x;
        my_matriz.y = y;
        
    }

    my_matriz.vetor = (float*) malloc(sizeof(float)* my_matriz.x * my_matriz.y);
    
    for (int i = 0 ; i <  my_matriz.x * my_matriz.y; i++ )
        fscanf(f, "%f",  &my_matriz.vetor[i]);
    fclose(f);
    hipMalloc(&(my_matriz.cuda_vetor), sizeof(float)* my_matriz.x * my_matriz.y );
    hipMemcpy(my_matriz.cuda_vetor,my_matriz.vetor,sizeof(float) * my_matriz.x * my_matriz.y, hipMemcpyHostToDevice);
    

    return my_matriz;
}

#include <math.h>

__device__
float sigmoid(float x) {
     return 1 / (1 + exp(-x));
}





__global__
void mm_kernel(float *mat1, float *mat2, float *mat3, int row1,int col1,int row2,int col2){  
    
    __shared__ float mat1_local[BLOCK_DIM][BLOCK_DIM];
    __shared__ float mat2_local[BLOCK_DIM][BLOCK_DIM];
    float soma = 0;


    for( int local_idx_block = 0; local_idx_block < ceilf((float)col1/BLOCK_DIM);local_idx_block++   ){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = local_idx_block * blockDim.y + threadIdx.y;
         
            
        if ((i <  row1 ) && (j < col1)){
            mat1_local[threadIdx.x][threadIdx.y] = mat1[i * col1 + j ];
        }
        else{
            mat1_local[threadIdx.x][threadIdx.y] = 0 ;
        }

        i = local_idx_block * blockDim.x + threadIdx.x;
        j = blockIdx.y * blockDim.y + threadIdx.y;
        if ((i <  row2 ) && (j < col2)){
            mat2_local[threadIdx.x][threadIdx.y] = mat2[i * col2 + j ];
        }
        else{
            mat2_local[threadIdx.x][threadIdx.y] = 0 ;
        }
        __syncthreads();
        for(int k = 0; k < BLOCK_DIM; k++){

            
           soma+= mat1_local[threadIdx.x][k] * mat2_local[k][threadIdx.y];
        }
        
         
        __syncthreads();
        
         

    }
    
         
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ((i < row1) && (j < col1))    {
        mat3[i *  col2+ j] = sigmoid(soma);
    }
        
   

}





int main(){
    TMATRIZ entrada = ler_matriz("input/entrada", 1);
    char file_name[] = "input/peso-0";
    
    
    TMATRIZ *pesos = (TMATRIZ*) malloc(sizeof(TMATRIZ) * entrada.num_de_pesos);
    for(int i = 0; i < entrada.num_de_pesos; i++){
        pesos[i] = ler_matriz(file_name, 0);
        file_name[11]+=1;
    }
    float * matriz_atual = entrada.cuda_vetor;
    int x_atual = entrada.x;
    int y_atual = entrada.y;

    dim3 threads_per_block(32, 32);
    dim3 blocks_per_grid(  1, 1);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0 ; i < entrada.num_de_pesos;i++){
        float *temp;
        hipMalloc(&temp, sizeof(float) * x_atual * pesos[i].y);
        blocks_per_grid.x =  std::ceil(  (float)x_atual / 32);
        blocks_per_grid.y = std::ceil( (float)pesos[i].y / 32);
        mm_kernel<<<blocks_per_grid,threads_per_block>>>(matriz_atual, pesos[i].cuda_vetor, temp,x_atual, y_atual,y_atual, pesos[i].y);
        hipDeviceSynchronize();
        hipFree(matriz_atual);
        matriz_atual = temp;
        hipFree(pesos[i].cuda_vetor); 
        y_atual = pesos[i].y;
    }
    hipEventRecord(stop);
    float milisegundos;
    hipEventElapsedTime(&milisegundos, start,stop);
    FILE *f = fopen( "gpu_otimizado.output" ,"a");
    fprintf(f, "%f\n", milisegundos);
    fclose(f);
    

    hipFree(matriz_atual);
    





    
    
}